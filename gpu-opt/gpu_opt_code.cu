#include "hip/hip_runtime.h"
// BSD 3-Clause License
// 
// Copyright (c) 2019, Karan Aggarwal (karan@iisc.ac.in)
// All rights reserved.
// 
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
// 
// 1. Redistributions of source code must retain the above copyright notice, this
//    list of conditions and the following disclaimer.
// 
// 2. Redistributions in binary form must reproduce the above copyright notice,
//    this list of conditions and the following disclaimer in the documentation
//    and/or other materials provided with the distribution.
// 
// 3. Neither the name of the copyright holder nor the names of its
//    contributors may be used to endorse or promote products derived from
//    this software without specific prior written permission.
// 
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
// OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/types.h>
#define FULL_MASK 0xffffffff

#if __CUDA_ARCH__ < 600
__device__ double atomicAdd(double *address, double val) {
  unsigned long long *address_as_ull = (unsigned long long *)address;
  unsigned long long old = *address_as_ull, assumed;

  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed,
                    __double_as_longlong(val + __longlong_as_double(assumed)));

  } while (assumed != old);

  return __longlong_as_double(old);
}
#endif

__global__ void compute_diag_sub(double *dPtr, const unsigned long *atomsPtr,
                                 const unsigned long *fibersPtr,
                                 const double *valuesPtr, const double *DPtr,
                                 const unsigned long nFibers, const int nTheta,
                                 const unsigned long nCoeffs) {

  unsigned long k = threadIdx.x + blockIdx.x * blockDim.x;
  unsigned long offset = 0;
  unsigned long stride = gridDim.x * blockDim.x;
  while ((k + offset) < nCoeffs) {
    double val = 0;
    int atom_index = atomsPtr[k + offset];
    for (int i = 0; i < nTheta; i++) {
      val += DPtr[atom_index + i] * DPtr[atom_index + i];
    }
    val = val * valuesPtr[k + offset] * valuesPtr[k + offset];
    atomicAdd(&dPtr[fibersPtr[k + offset]], val);
    offset += stride;
  }
  return;
}

__global__ void M_times_w(double *YPtr, const unsigned long *atomsPtr,
                          const unsigned long *voxelsPtr,
                          const unsigned long *fibersPtr,
                          const double *valuesPtr, const double *DPtr,
                          const double *wPtr, const int nTheta,
                          const unsigned long nVoxels,
                          const unsigned long nCoeffs, const unsigned long *vox,
                          const long nvox, int ch) {
  unsigned long long k = (threadIdx.x / 32) + (blockIdx.x * nc_mw);
  if (k < nvox) {
    if (ch == 0) {
      unsigned long voxel_index = voxelsPtr[vox[k]];
      __shared__ double y[nc_mw][Theta];
      int th_id = threadIdx.x % 32;
      while (th_id < nTheta) {
        y[threadIdx.x / 32][th_id] = YPtr[voxel_index + th_id];
        th_id = th_id + 32;
      }
      __syncwarp();
#pragma unroll 8
      for (int t = vox[k]; t < vox[k + 1]; t++) {
        unsigned long fiber_index = fibersPtr[t];
        unsigned long atom_index = atomsPtr[t];
        if (wPtr[fiber_index]) {
          th_id = threadIdx.x % 32;
          double val = wPtr[fiber_index] * valuesPtr[t];
          while (th_id < nTheta) {
            y[threadIdx.x / 32][th_id] += DPtr[atom_index + th_id] * val;
            th_id = th_id + 32;
          }
        }
        __syncwarp();
      }
      __syncwarp();
      th_id = threadIdx.x % 32;
      while (th_id < nTheta) {
        YPtr[voxel_index + th_id] = y[threadIdx.x / 32][th_id];
        th_id = th_id + 32;
      }
    } else {
      unsigned long voxel_index = voxelsPtr[k];
      unsigned long fiber_index = fibersPtr[k];
      unsigned long atom_index = atomsPtr[k];

      int th_id = threadIdx.x % 32;
      if (wPtr[fiber_index]) {
        double val = wPtr[fiber_index] * valuesPtr[k];
        while (th_id < nTheta) {
          atomicAdd(&YPtr[voxel_index + th_id], DPtr[atom_index + th_id] * val);
          th_id = th_id + 32;
        }
      }
    }
  }
  return;
}

__global__ void Mtransp_times_b(
    double *wPtr, const unsigned long *atomsPtr, const unsigned long *voxelsPtr,
    const unsigned long *fibersPtr, const double *valuesPtr, const double *DPtr,
    const double *YPtr, const unsigned long nFibers, const int nTheta,
    const long nCoeffs, const unsigned long *vox, const long nvox, int ch) {
  unsigned long long k = (threadIdx.x / 32) + (blockIdx.x * nc_my);
  if (k < nvox) {
    if (ch == 0) {
      for (int t = vox[k]; t < vox[k + 1]; t++) {
        unsigned long voxel_index = voxelsPtr[t];
        unsigned long atom_index = atomsPtr[t];
        unsigned long fiber_index = fibersPtr[t];

        double val = 0;
        int th_id = threadIdx.x % 32;
        while (th_id < nTheta) {
          val = val + (DPtr[atom_index + th_id] * YPtr[voxel_index + th_id]);
          th_id = th_id + 32;
        }
        __syncwarp();
#pragma unroll 5
        for (int j = 16; j >= 1; j = j / 2) {
          val += __shfl_down_sync(FULL_MASK, val, j);
        }
        __syncwarp();
        if ((threadIdx.x % 32) == 0) {
          atomicAdd(&wPtr[fiber_index], val * valuesPtr[t]);
        }
        __syncwarp();
      }
    } else {
      unsigned long voxel_index = voxelsPtr[k];
      unsigned long atom_index = atomsPtr[k];
      unsigned long fiber_index = fibersPtr[k];

      double val = 0;
      int th_id = threadIdx.x % 32;
      while (th_id < nTheta) {
        val = val + (DPtr[atom_index + th_id] * YPtr[voxel_index + th_id]);
        th_id = th_id + 32;
      }
      __syncwarp();
#pragma unroll 5
      for (int j = 16; j >= 1; j = j / 2) {
        val += __shfl_down_sync(FULL_MASK, val, j);
      }
      __syncwarp();
      if ((threadIdx.x % 32) == 0) {
        atomicAdd(&wPtr[fiber_index], val * valuesPtr[k]);
      }
      __syncwarp();
    }
  }
  return;
}